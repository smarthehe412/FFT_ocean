#include "cuFFT.h"

#include <iostream>

cuFFT::cuFFT(unsigned int size) : N(size) {
    int rank = 1;
    int n[] = {N};
    int inembed_row[] = {N, N};
    int istride_row = 1, idist_row = N;
    hipfftPlanMany(&plan_row, rank, n,
                    inembed_row, istride_row, idist_row,
                    inembed_row, istride_row, idist_row,
                    HIPFFT_C2C, N);

    int inembed_col[] = {N, N};
    int istride_col = N, idist_col = 1;
    hipfftPlanMany(&plan_col, rank, n,
                    inembed_col, istride_col, idist_col,
                    inembed_col, istride_col, idist_col,
                    HIPFFT_C2C, N);

    hipMalloc(&d_buffer, 2 * N * N * sizeof(hipfftComplex));
}

cuFFT::~cuFFT() {
    hipfftDestroy(plan_row);
    hipfftDestroy(plan_col);
    hipFree(d_buffer);
}

void cuFFT::batch_fft(complex* h_data, bool is_row) {
    hipMemcpy(d_buffer, h_data, 2*N*N*sizeof(hipfftComplex), hipMemcpyHostToDevice);

    if(is_row) {
        hipfftExecC2C(plan_row, d_buffer, d_buffer, HIPFFT_FORWARD);
    } else {
        hipfftExecC2C(plan_col, d_buffer, d_buffer, HIPFFT_FORWARD);
    }

    hipMemcpy(h_data, d_buffer, 2*N*N*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
}