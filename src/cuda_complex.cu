#include "cuda_complex.h"

hipfftComplex complex_to_cufft(const complex& c) {
    return make_hipComplex(c.a, c.b);
}

complex cufft_to_complex(const hipfftComplex& c) {
    return complex(c.x, c.y);
}