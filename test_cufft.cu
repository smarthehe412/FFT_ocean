#include <stdlib.h>
#include <stdio.h>
#include <iostream>
 
#include <string.h>
#include <math.h>
 
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "src/fft.h"
#include "src/complex.h"
#define Ndim 2
#define NX 8
#define NY 4
 
 
void testplanmany() {
 
	int N[2];
	N[0] = NX, N[1] = NY;
	int NXY = N[0] * N[1];
    complex in[NX*NY];

	hipfftComplex *input = (hipfftComplex*) malloc(NXY * sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*) malloc(NXY * sizeof(hipfftComplex));
	int i;
	for (i = 0; i < NXY; i++) {
		in[i].a = input[i].x = i % 1000;
		in[i].b = input[i].y = 0;
	}
	hipfftComplex *d_inputData, *d_outData;
	hipMalloc((void**) &d_inputData, N[0] * N[1] * sizeof(hipfftComplex));
	hipMalloc((void**) &d_outData, N[0] * N[1] * sizeof(hipfftComplex));
	hipMemcpy(d_inputData, input, N[0] * N[1] * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    std::cerr <<"check "<< sizeof(d_inputData) <<std::endl;
	hipfftHandle plan;
	/*
	hipfftMakePlanMany(hipfftHandle plan, int rank, int *n, int *inembed,
	int istride, int idist, int *onembed, int ostride,
	int odist, hipfftType type, int batch, size_t *workSize);
	 */
	int rank=1;
	int n[1];
	n[0]=NX;
	int istride=1;
	int idist = NX;
	int ostride=1;
	int odist = NX;
	int inembed[2];
	int onembed[2];
	inembed[0]=NX;  onembed[0]=NX;
	inembed[1] = NY; onembed[0] = NY;
 
	hipfftPlanMany(&plan,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_C2C, NY);
	hipfftExecC2C(plan, d_inputData, d_outData, HIPFFT_FORWARD);
	hipMemcpy(output, d_outData, NXY * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
 
	for (i = 0; i < NXY; i++) {
		if(i%NX==0)
			std::cerr << std::endl;
		std::cerr << output[i].x << " " << output[i].y << std::endl;
	}

    // for (int i = 0; i < NXY; i++) std::cout <<in[i].a <<' '<< in[i].b<<std::endl;

    cFFT fft(NX);
    for (i = 0; i < NY; i++) {
			fft.fft(in, in, 1, i * NX);
    }

    for (i = 0; i < NXY; i++) {
		if(i%NX==0)
			std::cerr << std::endl;
		std::cerr << in[i].a << " " << in[i].b << std::endl;
	}
 
	hipfftDestroy(plan);
	free(input);
	free(output);
	hipFree(d_inputData);
	hipFree(d_outData);
}
 
int main() {
 
	testplanmany();
}