
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
 
int main() {
    hipfftHandle plan;
    hipfftPlan1d(&plan, 1024, HIPFFT_Z2Z, 1); // 创建一个1D FFT计划
    hipfftDestroy(plan); // 销毁计划
    std::cout << "cuFFT initialized successfully." << std::endl;
    return 0;
}